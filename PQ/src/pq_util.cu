#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/pq_util.cuh"

__global__ void getAsymmetricDistanceKernel(float *distance, float *distance_tab, unsigned int *index, unsigned int size)
{
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = bid * blockDim.x + tid;

    if (idx < size)
    {
        distance[idx] = distance_tab[index[idx]];
    }
}

void cudaGetAsymmetricDistance(float *distance, float *distance_tab, unsigned int *index, const unsigned int size)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    size_t distance_bytes = size * sizeof(float);
    size_t tab_bytes = K * sizeof(float);
    size_t index_bytes = size * sizeof(unsigned int);

    float *d_distance, *d_distance_tab;
    unsigned int *d_index;
    hipMalloc((void **)&d_distance, distance_bytes);
    hipMalloc((void **)&d_distance_tab, tab_bytes);
    hipMalloc((void **)&d_index, index_bytes);

    hipMemcpy(d_distance, distance, distance_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_distance_tab, distance_tab, tab_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_index, index, index_bytes, hipMemcpyHostToDevice);

    dim3 block(1024);
    unsigned int grid_dim = ceil(sqrt((size + block.x - 1) / block.x));
    dim3 grid(grid_dim, grid_dim);
    getAsymmetricDistanceKernel<<<grid, block>>>(d_distance, d_distance_tab, d_index, size);

    hipMemcpy(distance, d_distance, distance_bytes, hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    hipFree(d_distance);
    hipFree(d_distance_tab);
    hipFree(d_index);
}