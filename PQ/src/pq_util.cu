#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/pq_util.cuh"

__global__ void getAsymmetricDistanceKernel(float *distance, float *distance_tab, size_t *index, size_t size)
{
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int idx = bid * blockDim.x + tid;

    if (idx < size)
    {
        distance[idx] = distance_tab[index[idx]];
    }
}

void cudaGetAsymmetricDistance(float *distance, float *distance_tab, size_t *index, const size_t size)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    size_t distance_bytes = size * sizeof(float);
    size_t tab_bytes = K * sizeof(float);
    size_t index_bytes = size * sizeof(size_t);

    float *d_distance, *d_distance_tab;
    size_t *d_index;
    hipMalloc((void **)&d_distance, distance_bytes);
    hipMalloc((void **)&d_distance_tab, tab_bytes);
    hipMalloc((void **)&d_index, index_bytes);

    hipMemcpy(d_distance, distance, distance_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_distance_tab, distance_tab, tab_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_index, index, index_bytes, hipMemcpyHostToDevice);

    dim3 block(1024);
    size_t grid_dim = ceil(sqrt((size + block.x - 1) / block.x));
    dim3 grid(grid_dim, grid_dim);
    getAsymmetricDistanceKernel<<<grid, block>>>(d_distance, d_distance_tab, d_index, size);

    hipMemcpy(distance, d_distance, distance_bytes, hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    hipFree(d_distance);
    hipFree(d_distance_tab);
    hipFree(d_index);
}