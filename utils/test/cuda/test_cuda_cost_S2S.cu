#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cstring>
#include "../../include/common.h"
#include "../../include/common.cuh"
#include "../../include/config.h"

int main(int argc, char const *argv[])
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distrib(0, 1);

    float *S1, *S2;
    S1 = (float *)malloc(TEST_SIZE * TEST_DIM * sizeof(float));
    S2 = (float *)malloc(TEST_SIZE * TEST_DIM * sizeof(float));
    for (int i = 0; i < TEST_SIZE * TEST_DIM; i++)
    {
        S1[i] = distrib(gen);
        S2[i] = distrib(gen);
    }

    float cost = costFromS2S(S1, S2, TEST_DIM, TEST_SIZE, TEST_SIZE);
    std::cout << "host finished" << std::endl;
    float cost_cuda = cudaCostFromS2S(S1, S2, TEST_DIM, TEST_SIZE, TEST_SIZE);

    free(S1);
    free(S2);

    if (abs(cost - cost_cuda) > 1e-3)
    {
        std::cout << "host: " << cost << ", cuda: " << cost_cuda << "\n";
        return -1;
    }

    return 0;
}
