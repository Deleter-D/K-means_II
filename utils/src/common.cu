#include "hip/hip_runtime.h"
#include "../include/common.cuh"
#include "../include/config.h"
#include <stdio.h>
#include <random>

__global__ void euclideanDistanceKernel(float *distance, float *vec, float *set, float *temp, const int dim, const int size)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    // 计算每对元素差的平方，存入temp
    if (idx < dim * size)
    {
        temp[idx] = (vec[tid] - set[idx]) * (vec[tid] - set[idx]);
    }
    __syncthreads();

    // 将每个向量的96个元素归约为32个元素
    __shared__ float s_data[32];
    if (tid < 32)
        s_data[tid] = temp[idx] + temp[idx + 32] + temp[idx + 64];
    __syncthreads();

    // 交错匹配的归约求和
    for (int stride = 16; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            s_data[tid] += s_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)
        distance[blockIdx.x] = s_data[0];
}

float *cudaEuclideanDistance(float *vec, float *set, const int dim, const int size)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    size_t distance_bytes = size * sizeof(float);
    size_t vec_bytes = dim * sizeof(float);
    size_t set_bytes = dim * size * sizeof(float);

    float *distance = (float *)malloc(distance_bytes);

    float *d_distance, *d_vec, *d_set, *temp;
    hipMalloc((void **)&d_distance, distance_bytes);
    hipMalloc((void **)&d_vec, vec_bytes);
    hipMalloc((void **)&d_set, set_bytes);
    hipMalloc((void **)&temp, set_bytes);

    hipMemcpy(d_vec, vec, vec_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_set, set, set_bytes, hipMemcpyHostToDevice);
    hipMemset(d_distance, 0, distance_bytes);
    hipMemset(temp, 0, set_bytes);

    dim3 block(dim);
    dim3 grid((size * dim + block.x - 1) / block.x);
    euclideanDistanceKernel<<<grid, block, 32 * sizeof(float), stream>>>(d_distance, d_vec, d_set, temp, dim, size);

    hipMemcpy(distance, d_distance, distance_bytes, hipMemcpyDeviceToHost);
    hipStreamSynchronize(stream);

    hipFree(d_distance);
    hipFree(d_vec);
    hipFree(d_set);
    hipFree(temp);

    hipStreamDestroy(stream);

    return distance;
}

float cudaCostFromV2S(float *vec, float *cluster_set, const int dim, const size_t size)
{
    float min = MAXFLOAT;

    float *distance = cudaEuclideanDistance(vec, cluster_set, dim, size);

    // TODO: 取最小值可优化
    for (size_t i = 0; i < size; i++)
    {
        if (distance[i] < min)
            min = distance[i];
    }

    return min;
}

float cudaCostFromS2S(float *original_set, float *cluster_set, const int dim, const size_t original_size, const size_t cluster_size)
{
    float *sums = (float *)malloc(original_size * sizeof(float));
    for (size_t i = 0; i < original_size; i++)
    {
        sums[i] = cudaCostFromV2S(&original_set[i * dim], cluster_set, dim, cluster_size);
    }
    hipDeviceSynchronize();

    float sum = 0.0f;
    for (size_t i = 0; i < original_size; i++)
    {
        sum += sums[i];
    }

    free(sums);

    return sum;
}

size_t cudaBelongV2S(float *x, float *cluster_set, const int dim, const size_t size)
{
    float min = MAXFLOAT;
    size_t index;

    float *distance = cudaEuclideanDistance(x, cluster_set, dim, size);

    // TODO: 取最小值可优化
    for (size_t i = 0; i < size; i++)
    {
        if (distance[i] < min)
        {
            min = distance[i];
            index = i;
        }
    }

    return index;
}

size_t *cudaBelongS2S(float *original_set, float *cluster_set, const int dim, const size_t original_size, const size_t cluster_size)
{
    size_t *index = (size_t *)malloc(original_size * sizeof(size_t));
    for (size_t i = 0; i < original_size; i++)
    {
        index[i] = cudaBelongV2S(&original_set[i * dim], cluster_set, dim, cluster_size);
    }
    hipDeviceSynchronize();
    return index;
}

float *cudaKmeanspp(float *cluster_set, size_t *omega, size_t k, const int dim, const size_t cluster_size)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> distrib(0, cluster_size - 1);

    size_t index = distrib(gen);

    // 申请最终聚类中心集的内存
    float *cluster_final = (float *)malloc(k * dim * sizeof(float));
    // 均匀分布中随机采样一个原聚类中心集的向量放入最终聚类中心集中
    memcpy(&cluster_final[0], &cluster_set[index * dim], dim * sizeof(float));
    size_t current_k = 1;

    float max_p;
    float temp_p;
    size_t max_p_index;

    // 迭代k-1次，每次取一个聚类中心进入c_final
    while (current_k < k)
    {
        max_p = -1.0f;
        float cost_set2final = cudaCostFromS2S(cluster_set, cluster_final, dim, cluster_size, current_k);
        hipDeviceSynchronize();
        for (size_t i = 0; i < cluster_size; i++)
        {
            // 计算当前向量的概率
            temp_p = omega[i] * cudaCostFromV2S(&cluster_set[i * dim], cluster_final, dim, current_k) / cost_set2final;
            hipDeviceSynchronize();
            // 记录概率最大的向量信息
            if (temp_p > max_p)
            {
                max_p = temp_p;
                max_p_index = i;
            }
        }
        // 将概率最大的向量并入最终聚类中心集
        memcpy(&cluster_final[current_k * dim], &cluster_set[max_p_index * dim], dim * sizeof(float));
        current_k++;
    }

    return cluster_final;
}

__global__ void getNewClusterKernel(float *cluster_new, float *original_set, size_t *belong, const int dim, const size_t original_size, unsigned int *count)
{
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    for (size_t i = tid, j = 0; j < original_size; i += dim, j++)
    {
        if (belong[j] == bid)
        {
            cluster_new[idx] += original_set[i];
            count[belong[j]]++;
        }
    }
    __syncthreads();

    cluster_new[idx] /= count[bid];
}

float *cudaGetNewCluster(float *original_set, size_t *belong, const int dim, const size_t original_size)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    size_t cluster_bytes = dim * K * sizeof(float);
    size_t origianl_set_bytes = dim * original_size * sizeof(float);
    size_t belong_bytes = original_size * sizeof(size_t);
    size_t count_bytes = K * sizeof(unsigned int);

    float *cluster_new = (float *)malloc(cluster_bytes);

    float *d_cluster_new, *d_original_set;
    size_t *d_belong;
    unsigned int *d_count;
    hipMalloc((void **)&d_cluster_new, cluster_bytes);
    hipMalloc((void **)&d_original_set, origianl_set_bytes);
    hipMalloc((void **)&d_belong, belong_bytes);
    hipMalloc((void **)&d_count, count_bytes);

    hipMemset(d_cluster_new, 0, cluster_bytes);
    hipMemcpy(d_original_set, original_set, origianl_set_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_belong, belong, belong_bytes, hipMemcpyHostToDevice);
    hipMemset(d_count, 0, count_bytes);

    dim3 block(dim);
    dim3 grid(K);
    getNewClusterKernel<<<grid, block>>>(d_cluster_new, d_original_set, d_belong, dim, original_size, d_count);

    hipMemcpy(cluster_new, d_cluster_new, cluster_bytes, hipMemcpyDeviceToHost);
    hipStreamSynchronize(stream);

    hipFree(d_cluster_new);
    hipFree(d_original_set);
    hipFree(d_belong);
    hipFree(d_count);

    hipStreamDestroy(stream);

    return cluster_new;
}