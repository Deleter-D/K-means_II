#include "hip/hip_runtime.h"
#include "../include/common.cuh"
#include "../include/config.h"
#include <stdio.h>
#include <random>
#include <omp.h>
#include <math.h>
#include <iostream>

__global__ void euclideanDistanceKernel(float *distance, float *vec, float *set, float *temp, const int dim, const int size)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    // 计算每对元素差的平方，存入temp
    if (idx < dim * size)
    {
        temp[idx] = (vec[tid] - set[idx]) * (vec[tid] - set[idx]);
    }
    __syncthreads();

    extern __shared__ float s_data[];
    s_data[tid] = temp[idx];
    // if (tid < 32)
    //     s_data[tid] = temp[idx] + temp[idx + 32] + temp[idx + 64];
    __syncthreads();

    // 交错匹配的归约求和
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            s_data[tid] += s_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)
        distance[blockIdx.x] = s_data[0];
}

void cudaEuclideanDistance(float *distance, float *vec, float *set, const int dim, const int size)
{
    size_t MAX_SIZE = 8e9 / (dim * sizeof(float));
    int iter_times = (size / MAX_SIZE) + 1;
    int size_per_iter;
    int size_last_iter;
    if (iter_times == 1)
    {
        size_per_iter = size;
        size_last_iter = size;
    }
    else
    {
        size_per_iter = MAX_SIZE;
        size_last_iter = size - MAX_SIZE * (iter_times - 1);
    }

    hipStream_t *stream = (hipStream_t *)malloc(iter_times * sizeof(hipStream_t));
    size_t vec_bytes = dim * sizeof(float);
    float **d_distances, **d_vecs, **d_sets, **temps;
    d_distances = (float **)malloc(iter_times * sizeof(float *));
    d_vecs = (float **)malloc(iter_times * sizeof(float *));
    d_sets = (float **)malloc(iter_times * sizeof(float *));
    temps = (float **)malloc(iter_times * sizeof(float *));

    for (int i = 0; i < iter_times; i++)
    {
        hipStreamCreate(&stream[i]);

        size_t distance_bytes;
        size_t set_bytes;
        size_t current_size;

        if (i == iter_times - 1)
        {
            distance_bytes = size_last_iter * sizeof(float);
            set_bytes = dim * size_last_iter * sizeof(float);
            current_size = size_last_iter;
        }
        else
        {
            distance_bytes = size_per_iter * sizeof(float);
            set_bytes = dim * size_per_iter * sizeof(float);
            current_size = size_per_iter;
        }

        // float *d_distance, *d_vec, *d_set, *temp;
        hipMalloc((void **)&d_distances[i], distance_bytes);
        hipMalloc((void **)&d_vecs[i], vec_bytes);
        hipMalloc((void **)&d_sets[i], set_bytes);
        hipMalloc((void **)&temps[i], set_bytes);

        hipMemcpy(d_vecs[i], vec, vec_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_sets[i], &set[i * dim * size_per_iter], set_bytes, hipMemcpyHostToDevice);
        hipMemset(d_distances[i], 0, distance_bytes);
        hipMemset(temps[i], 0, set_bytes);

        dim3 block(dim);
        dim3 grid((current_size * dim + block.x - 1) / block.x);
        euclideanDistanceKernel<<<grid, block, dim * sizeof(float), stream[i]>>>(d_distances[i], d_vecs[i], d_sets[i], temps[i], dim, current_size);

        hipMemcpy(&distance[i * size_per_iter], d_distances[i], distance_bytes, hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < iter_times; i++)
    {
        hipStreamSynchronize(stream[i]);

        hipFree(d_distances[i]);
        hipFree(d_vecs[i]);
        hipFree(d_sets[i]);
        hipFree(temps[i]);

        hipStreamDestroy(stream[i]);
    }
    free(stream);
    free(d_distances);
    free(d_vecs);
    free(d_sets);
    free(temps);
}

float cudaCostFromV2S(float *vec, float *cluster_set, const int dim, const size_t size)
{
    float min = 3.40282347e+38F;

    float *distance = (float *)malloc(size * sizeof(float));
    cudaEuclideanDistance(distance, vec, cluster_set, dim, size);

#pragma omp parallel for private(i) reduction(min : min)
    for (size_t i = 0; i < size; i++)
    {
        if (distance[i] < min)
            min = distance[i];
    }

    free(distance);

    return min;
}

__global__ void costFromS2SKernel(float *distances, float *original_set, float *cluster_set, int dim, size_t original_size, size_t cluster_size)
{
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float distance_temp[];

    if (idx < original_size)
    {
        distances[idx] = INFINITY;

        for (int j = 0; j < cluster_size; j++)
        {
            float dist = 0.0f;
            for (int k = 0; k < dim; k++)
            {
                float diff = original_set[idx * dim + k] - cluster_set[j * dim + k];
                dist += diff * diff;
            }
            distance_temp[tid] = dist;

            if (distance_temp[tid] < distances[idx])
            {
                distances[idx] = distance_temp[tid];
            }
        }
    }
}

float cudaCostFromS2S(float *original_set, float *cluster_set, const int dim, const size_t original_size, const size_t cluster_size)
{
    float *distances = (float *)malloc(original_size * sizeof(float));

    size_t MAX_SIZE = 8e9 / (dim * sizeof(float));
    int iter_times = (original_size / MAX_SIZE) + 1;
    int size_per_iter;
    int size_last_iter;
    if (iter_times == 1)
    {
        size_per_iter = original_size;
        size_last_iter = original_size;
    }
    else
    {
        size_per_iter = MAX_SIZE;
        size_last_iter = original_size - MAX_SIZE * (iter_times - 1);
    }

    hipStream_t *stream = (hipStream_t *)malloc(iter_times * sizeof(hipStream_t));
    float **d_original_sets, **d_distances;
    d_original_sets = (float **)malloc(iter_times * sizeof(float *));
    d_distances = (float **)malloc(iter_times * sizeof(float *));

    float *d_cluster_set;
    hipMalloc((void **)&d_cluster_set, cluster_size * dim * sizeof(float));
    hipMemcpy(d_cluster_set, cluster_set, cluster_size * dim * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < iter_times; i++)
    {
        hipStreamCreate(&stream[i]);

        size_t distance_bytes;
        size_t set_bytes;
        size_t current_size;

        if (i == iter_times - 1)
        {
            distance_bytes = size_last_iter * sizeof(float);
            set_bytes = dim * size_last_iter * sizeof(float);
            current_size = size_last_iter;
        }
        else
        {
            distance_bytes = size_per_iter * sizeof(float);
            set_bytes = dim * size_per_iter * sizeof(float);
            current_size = size_per_iter;
        }

        hipMalloc((void **)&d_original_sets[i], set_bytes);
        hipMalloc((void **)&d_distances[i], distance_bytes);

        hipMemcpy(d_original_sets[i], &original_set[i * dim * size_per_iter], set_bytes, hipMemcpyHostToDevice);
        hipMemset(d_distances[i], 0, distance_bytes);

        dim3 block(1024);
        dim3 grid((current_size + block.x - 1) / block.x);

        costFromS2SKernel<<<grid, block, block.x * sizeof(float), stream[i]>>>(d_distances[i], d_original_sets[i], d_cluster_set, dim, current_size, cluster_size);

        hipMemcpy(&distances[i * size_per_iter], d_distances[i], distance_bytes, hipMemcpyDeviceToHost);

        auto err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cout << hipGetErrorString(err) << "\n";
        }
    }

    for (int i = 0; i < iter_times; i++)
    {
        hipStreamSynchronize(stream[i]);

        hipFree(d_original_sets[i]);
        hipFree(d_distances[i]);

        hipStreamDestroy(stream[i]);
    }
    hipFree(d_cluster_set);
    free(stream);
    free(d_original_sets);
    free(d_distances);

    float sum = 0.0f;
#pragma omp parallel for reduction(+ : sum)
    for (size_t i = 0; i < original_size; i++)
    {
        sum += distances[i];
    }

    free(distances);

    return sum;
}

size_t cudaBelongV2S(float *x, float *cluster_set, const int dim, const size_t size)
{
    float min = 3.40282347e+38F;
    size_t index;

    float *distance = (float *)malloc(size * sizeof(float));
    cudaEuclideanDistance(distance, x, cluster_set, dim, size);

#pragma omp parallel for private(i) reduction(min : min)
    for (size_t i = 0; i < size; i++)
    {
        if (distance[i] < min)
        {
            min = distance[i];
            index = i;
        }
    }

    free(distance);

    return index;
}

__global__ void belongS2SKernel(size_t *indices, float *distances, float *original_set, float *cluster_set, int dim, size_t original_size, size_t cluster_size)
{
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float distance_temp[];

    if (idx < original_size)
    {
        distances[idx] = INFINITY;

        for (int i = 0; i < cluster_size; i++)
        {
            float dist = 0.0f;
            for (int j = 0; j < dim; j++)
            {
                float diff = original_set[idx * dim + j] - cluster_set[i * dim + j];
                dist += diff * diff;
            }
            distance_temp[tid] = dist;

            if (distance_temp[tid] < distances[idx])
            {
                distances[idx] = distance_temp[tid];

                indices[idx] = i;
            }
        }
    }
}

void cudaBelongS2S(size_t *index, float *original_set, float *cluster_set, const int dim, const size_t original_size, const size_t cluster_size)
{
    size_t MAX_SIZE = 8e9 / (dim * sizeof(float));
    int iter_times = (original_size / MAX_SIZE) + 1;
    int size_per_iter;
    int size_last_iter;
    if (iter_times == 1)
    {
        size_per_iter = original_size;
        size_last_iter = original_size;
    }
    else
    {
        size_per_iter = MAX_SIZE;
        size_last_iter = original_size - MAX_SIZE * (iter_times - 1);
    }

    hipStream_t *stream = (hipStream_t *)malloc(iter_times * sizeof(hipStream_t));
    float **d_original_sets, **d_distances;
    size_t **d_indices;
    d_original_sets = (float **)malloc(iter_times * sizeof(float *));
    d_distances = (float **)malloc(iter_times * sizeof(float *));
    d_indices = (size_t **)malloc(iter_times * sizeof(size_t *));

    float *d_cluster_set;
    hipMalloc((void **)&d_cluster_set, cluster_size * dim * sizeof(float));
    hipMemcpy(d_cluster_set, cluster_set, cluster_size * dim * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < iter_times; i++)
    {
        hipStreamCreate(&stream[i]);

        size_t distance_bytes;
        size_t index_bytes;
        size_t set_bytes;

        if (i == iter_times - 1)
        {
            distance_bytes = size_last_iter * sizeof(float);
            index_bytes = size_last_iter * sizeof(size_t);
            set_bytes = dim * size_last_iter * sizeof(float);
        }
        else
        {
            distance_bytes = size_per_iter * sizeof(float);
            index_bytes = size_per_iter * sizeof(size_t);
            set_bytes = dim * size_per_iter * sizeof(float);
        }

        hipMalloc((void **)&d_original_sets[i], set_bytes);
        hipMalloc((void **)&d_distances[i], distance_bytes);
        hipMalloc((void **)&d_indices[i], index_bytes);

        hipMemcpy(d_original_sets[i], &original_set[i * dim * size_per_iter], set_bytes, hipMemcpyHostToDevice);

        dim3 block(1024);
        size_t current_size;
        if (i == iter_times - 1)
        {
            current_size = size_last_iter;
        }
        else
        {
            current_size = size_per_iter;
        }
        dim3 grid((current_size + block.x - 1) / block.x);

#ifdef DEBUG
        std::cout << "The " << i << "th iteration of thread " << omp_get_thread_num() << " is workding.\n";
#endif

        belongS2SKernel<<<grid, block, block.x * sizeof(float), stream[i]>>>(d_indices[i], d_distances[i], d_original_sets[i], d_cluster_set, dim, current_size, cluster_size);

        hipMemcpy(&index[i * size_per_iter], d_indices[i], index_bytes, hipMemcpyDeviceToHost);

        auto err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cout << hipGetErrorString(err) << "\n";
        }
    }

    for (int i = 0; i < iter_times; i++)
    {
        hipStreamSynchronize(stream[i]);

        hipFree(d_original_sets[i]);
        hipFree(d_distances[i]);
        hipFree(d_indices[i]);

        hipStreamDestroy(stream[i]);
    }
    hipFree(d_cluster_set);
    free(stream);
    free(d_original_sets);
    free(d_distances);
    free(d_indices);
}

__global__ void kmeansppKernel(size_t *indices, float *probability, float *cluster_set, float *cluster_final, size_t *omega, int dim, int current_k, int cluster_size)
{
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    extern __shared__ float p_temp[];

    if (idx < cluster_size)
    {
        probability[idx] = INFINITY;

        for (int i = 0; i < current_k; i++)
        {
            p_temp[tid] = 0.0f;
            for (int j = 0; j < dim; j++)
            {
                float diff = cluster_set[idx * dim + j] - cluster_final[i * dim + j];
                p_temp[tid] += diff * diff;
            }
            p_temp[tid] *= omega[idx];

            if (p_temp[tid] < probability[idx])
            {
                probability[idx] = p_temp[tid];
                indices[idx] = i;
            }
        }
    }
}

void cudaKmeanspp(float *cluster_final, float *cluster_set, size_t *omega, size_t k, const int dim, const size_t cluster_size)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> distrib(0, cluster_size - 1);

    size_t index = distrib(gen);

    // 均匀分布中随机采样一个原聚类中心集的向量放入最终聚类中心集中
    memcpy(&cluster_final[0], &cluster_set[index * dim], dim * sizeof(float));
    size_t current_k = 1;

    float max_p;
    size_t max_p_index;

    size_t indices_bytes = cluster_size * sizeof(size_t);
    size_t cluster_set_bytes = cluster_size * dim * sizeof(float);
    size_t p_bytes = cluster_size * sizeof(float);

    size_t *indices = (size_t *)malloc(indices_bytes);
    float *probability = (float *)malloc(p_bytes);

    size_t *d_omega, *d_indices;
    float *d_cluster_set, *d_probability;
    hipMalloc((void **)&d_cluster_set, cluster_set_bytes);
    hipMalloc((void **)&d_omega, indices_bytes);
    hipMalloc((void **)&d_indices, indices_bytes);
    hipMalloc((void **)&d_probability, p_bytes);

    hipMemcpy(d_cluster_set, cluster_set, cluster_set_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_omega, omega, indices_bytes, hipMemcpyHostToDevice);

    // 迭代k-1次，每次取一个聚类中心进入c_final
    while (current_k < k)
    {
        max_p = -1.0f;

        hipStream_t stream;
        hipStreamCreate(&stream);

        size_t cluster_final_bytes = current_k * dim * sizeof(float);

        float *d_cluster_final;
        hipMalloc((void **)&d_cluster_final, cluster_final_bytes);

        hipMemcpy(d_cluster_final, cluster_final, cluster_final_bytes, hipMemcpyHostToDevice);

        dim3 block(1024);
        dim3 grid((cluster_size + block.x - 1) / block.x);
        kmeansppKernel<<<grid, block, block.x * sizeof(float), stream>>>(d_indices, d_probability, d_cluster_set, d_cluster_final, d_omega, dim, current_k, cluster_size);

        hipMemcpy(probability, d_probability, p_bytes, hipMemcpyDeviceToHost);
        hipMemcpy(indices, d_indices, indices_bytes, hipMemcpyDeviceToHost);

        hipStreamSynchronize(stream);
        hipFree(d_cluster_final);

#pragma omp parallel for private(i) reduction(max : max_p)
        for (int i = 0; i < cluster_size; i++)
        {
            if (probability[i] > max_p)
            {
                max_p = probability[i];
                max_p_index = indices[i];
            }
        }

        // 将概率最大的向量并入最终聚类中心集
        memcpy(&cluster_final[current_k * dim], &cluster_set[max_p_index * dim], dim * sizeof(float));
        current_k++;
    }

    hipFree(d_cluster_set);
    hipFree(d_omega);
    hipFree(d_indices);
    hipFree(d_probability);

    free(probability);
    free(indices);
}

__global__ void getNewClusterKernel(float *cluster_new, float *original_set, size_t *belong, const int dim, const size_t original_size, unsigned int *count)
{
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    extern __shared__ float sum[];
    __shared__ size_t count_current_blk;
    if (tid < dim)
        sum[tid] = 0.0f;
    if (tid == 0)
        count_current_blk = 0;
    __syncthreads();

    for (size_t i = tid, j = 0; j < original_size; i += dim, j++)
    {
        if (belong[j] == bid)
        {
            sum[tid] += original_set[i];
        }
        if (belong[j] == bid && tid == 0)
        {
            count_current_blk++;
        }
    }
    __syncthreads();

    if (tid < dim)
        cluster_new[idx] = sum[tid];

    if (tid == 0)
        count[bid] = count_current_blk;
    __syncthreads();

    if (idx < K * dim)
        cluster_new[idx] /= count[bid];
}

void cudaGetNewCluster(float *cluster_new, float *original_set, size_t *belong, const int dim, const size_t original_size)
{
    size_t cluster_bytes = dim * K * sizeof(float);
    size_t origianl_set_bytes = dim * original_size * sizeof(float);
    size_t belong_bytes = original_size * sizeof(size_t);
    size_t count_bytes = K * sizeof(unsigned int);

    float *d_cluster_new, *d_original_set;
    size_t *d_belong;
    unsigned int *d_count;
    hipMalloc((void **)&d_cluster_new, cluster_bytes);
    hipMalloc((void **)&d_original_set, origianl_set_bytes);
    hipMalloc((void **)&d_belong, belong_bytes);
    hipMalloc((void **)&d_count, count_bytes);

    hipMemset(d_cluster_new, 0, cluster_bytes);
    hipMemcpy(d_original_set, original_set, origianl_set_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_belong, belong, belong_bytes, hipMemcpyHostToDevice);
    hipMemset(d_count, 0, count_bytes);

    dim3 block(dim);
    dim3 grid(K);
    getNewClusterKernel<<<grid, block, (dim) * sizeof(float)>>>(d_cluster_new, d_original_set, d_belong, dim, original_size, d_count);

    hipMemcpy(cluster_new, d_cluster_new, cluster_bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_cluster_new);
    hipFree(d_original_set);
    hipFree(d_belong);
    hipFree(d_count);
}

__global__ void isCloseKernel(float *distance, float *cluster_new, float *cluster_old, float *temp, const int dim, const size_t cluster_size)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    // 计算每对元素差的平方，存入temp
    if (idx < dim * cluster_size)
    {
        temp[idx] = (cluster_new[idx] - cluster_old[idx]) * (cluster_new[idx] - cluster_old[idx]);
    }
    __syncthreads();

    extern __shared__ float s_data[];
    s_data[tid] = temp[idx];
    __syncthreads();

    // 交错匹配的归约求和
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            s_data[tid] += s_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)
        distance[blockIdx.x] = s_data[0];
}

bool cudaIsClose(float *cluster_new, float *cluster_old, const int dim, const size_t cluster_size, float epsilon)
{
    size_t distance_bytes = cluster_size * sizeof(float);
    size_t cluster_bytes = dim * cluster_size * sizeof(float);

    float *distance = (float *)malloc(distance_bytes);

    float *d_distance, *d_cluster_new, *d_cluster_old, *temp;
    hipMalloc((void **)&d_distance, distance_bytes);
    hipMalloc((void **)&d_cluster_new, cluster_bytes);
    hipMalloc((void **)&d_cluster_old, cluster_bytes);
    hipMalloc((void **)&temp, cluster_bytes);

    hipMemcpy(d_cluster_new, cluster_new, cluster_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_cluster_old, cluster_old, cluster_bytes, hipMemcpyHostToDevice);
    hipMemset(d_distance, 0, distance_bytes);
    hipMemset(temp, 0, cluster_bytes);

    dim3 block(dim);
    dim3 grid((cluster_size * dim + block.x - 1) / block.x);
    isCloseKernel<<<grid, block, dim * sizeof(float)>>>(d_distance, d_cluster_new, d_cluster_old, temp, dim, cluster_size);

    hipMemcpy(distance, d_distance, distance_bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_distance);
    hipFree(d_cluster_new);
    hipFree(d_cluster_old);
    hipFree(temp);

    for (size_t i = 0; i < cluster_size; i++)
    {
        if (distance[i] > epsilon)
        {
            free(distance);
            return false;
        }
    }

    free(distance);
    return true;
}